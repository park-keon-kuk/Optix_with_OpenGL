#include "hip/hip_runtime.h"
/*
* Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix_cuda.h>

using namespace optix;

// ����
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(float3, draw_color, , );

// ����
rtBuffer<float4, 2>   result_buffer;

// Ray Gen
RT_PROGRAM void draw_solid_color()
{
	size_t2 size = result_buffer.size();
	float3 color = make_float3(0, 0, 0);

	if (launch_index.x > 50 && launch_index.x < 100
		&& launch_index.y > 50 && launch_index.y < 100) 
	{
		color = make_float3(1, 0, 0);
	}

	result_buffer[launch_index] = make_float4(color, 0.f);
}

// Exception
RT_PROGRAM void exception()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y);
	result_buffer[launch_index] = make_float4(0.f, 0.f, 0.f, 0.f);
}